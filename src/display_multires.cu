
#include <stdio.h>
#include <stdlib.h>

#include <Exceptions.h>
#include <ImageIO.h>
#include <ImagesCPU.h>
#include <ImagesNPP.h>

#include <cassert>
#include <fstream>
#include <iostream>
#include <string.h>

#include <hip/hip_runtime.h>
#include <npp.h>

#include <hip/hip_runtime_api.h>
#include <helper_string.h>


struct Args {
    int k;
    char* input_name;
    char* output_name;
    int num_channels;
};

Args parse_args(int argc, char** argv) {
    if (argc!=5) {
        std::cerr << "Format:\n display_multires "
            "<NUM_RES> <INPUT_FILE> <OUTPUT_FILE> <NUMCHANNELS>\n";
        exit(-1);
    }
    Args args;
    //for (int ci=0; ci<argc; ci++) 
    //    std::cout << "arg " << ci << " " << std::string(argv[ci]) << std::endl;
    {
        args.k = atoi(argv[1]);
        assert(args.k>0 && args.k<7);
    }
    args.input_name = argv[2];
    args.output_name = argv[3];
    {
        args.num_channels = atoi(argv[4]);
        assert(args.num_channels==1 || args.num_channels==3);
    }
    return args;
}
    

int main (int argc, char **argv)
{

   Args args = parse_args(argc, argv); 


   exit (0);
}

