#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>

#include <Exceptions.h>
#include <ImageIO.h>
#include <ImagesCPU.h>
#include <ImagesNPP.h>

#include <cassert>
#include <fstream>
#include <iostream>
#include <string.h>

#include <hip/hip_runtime.h>
#include <npp.h>

#include <hip/hip_runtime_api.h>
#include <helper_string.h>

#define CUDA_CALL( call ) \
{\
     auto err = call;\
     if (hipSuccess !=err) {\
         printf("error %d %s in line %d", err, hipGetErrorName(err), __LINE__); \
         exit(-1); \
     } \
}

bool printfNPPinfo(int argc, char *argv[])
{
  const NppLibraryVersion *libVer = nppGetLibVersion();

  printf("NPP Library Version %d.%d.%d\n", libVer->major, libVer->minor,
         libVer->build);

  int driverVersion, runtimeVersion;
  hipDriverGetVersion(&driverVersion);
  hipRuntimeGetVersion(&runtimeVersion);

  printf("  CUDA Driver  Version: %d.%d\n", driverVersion / 1000,
         (driverVersion % 100) / 10);
  printf("  CUDA Runtime Version: %d.%d\n", runtimeVersion / 1000,
         (runtimeVersion % 100) / 10);

  // Min spec is SM 1.0 devices
  bool bVal = checkCudaCapabilities(1, 0);
  return bVal;
}


__host__ __device__ struct Rectangle {
    int x0;
    int y0;
    int width;
    int stride;
    int height;
    
    __host__ __device__ Rectangle(int x0, int y0, int width, int height, int pitch):
        x0(x0), y0(y0), width(width), height(height), stride(pitch) {}
    __host__ __device__ inline int x_end() {return x0 + width;}
    __host__ __device__ inline int x_stride_end() {return x0 + stride;}
    __host__ __device__ inline int y_end() {return y0 + height;}
    __host__ __device__ inline int numElements() {return height*stride;}
};


struct Args {
    int k;
    char* input_name;
    char* output_name;
    int num_channels;
};

Args parse_args(int argc, char** argv) {
    Args args;
    if (1) {
        args.k = 1;
        args.input_name = "data/sloth-gray.png";
        args.output_name = "data/sloth-gray-mr.png";
        return args;
    }

    if (argc!=4) {
        std::cerr << "Format:\n display_multires "
            "<NUM_RES> <INPUT_FILE> <OUTPUT_FILE>\n";
        exit(-1);
    }
    //for (int ci=0; ci<argc; ci++) 
    //    std::cout << "arg " << ci << " " << std::string(argv[ci]) << std::endl;
    {
        args.k = atoi(argv[1]);
        assert(args.k>0 && args.k<7);
    }
    args.input_name = argv[2];
    args.output_name = argv[3];
    return args;
}


#define X_STRIDE 64

// copy src image to first 2/3rds of image, fill the rest with zeros.
__global__ void init_trg_kernel(const Npp8u* src, Npp8u* trg,
        Rectangle srcRect, Rectangle trgRect) {
    const int tidx_x = gridDim.x * blockIdx.x + threadIdx.x;
    const int tidx_y = gridDim.y * blockIdx.y + threadIdx.y;

    const int src_addr = tidx_y * srcRect.stride + tidx_x;
    const int trg_addr = tidx_y * trgRect.stride + tidx_x;

    __shared__ Npp8u buff[X_STRIDE];

    if (tidx_x < srcRect.width && tidx_y < srcRect.height
            && src_addr < srcRect.numElements()) {
        buff[threadIdx.x] = src[src_addr];
    } else {
        buff[threadIdx.x] = 0;
    }
    __syncthreads();

    if (tidx_x < trgRect.stride && tidx_y < trgRect.height
            && trg_addr < trgRect.numElements()) {
        trg[trg_addr] = buff[threadIdx.x];
    }
}

int main (int argc, char **argv)
{

    hipDeviceReset();

    Args args = parse_args(argc, argv); 
    
    //findCudaDevice(argc, (const char **)argv);
    //gpuDeviceInit(0);
  
    const NppLibraryVersion *libVer = nppGetLibVersion();

    printf("NPP Library Version %d.%d.%d\n", libVer->major, libVer->minor,
             libVer->build);

    int driverVersion, runtimeVersion;
    hipDriverGetVersion(&driverVersion);
    hipRuntimeGetVersion(&runtimeVersion);

    printf("  CUDA Driver  Version: %d.%d\n", driverVersion / 1000,
            (driverVersion % 100) / 10);
    printf("  CUDA Runtime Version: %d.%d\n", runtimeVersion / 1000,
            (runtimeVersion % 100) / 10);

    checkCudaErrors(hipSetDevice(0));

    // declare a host image object for an 8-bit grayscale image
    npp::ImageCPU_8u_C1 h_src_C1;

    // load gray-scale image from disk, and create device image object.
    npp::loadImage(args.input_name, h_src_C1);
    npp::ImageNPP_8u_C1 d_src_C1(h_src_C1);
    
    Rectangle src_rect(0, 0, (int)d_src_C1.width(), (int)d_src_C1.height(),
            (int)d_src_C1.pitch());

    // The target image has 3/2 
    int trg_width = (src_rect.width*3+1) / 2;
    npp::ImageCPU_8u_C1 h_trg_C1(trg_width+1, src_rect.height);
    npp::ImageNPP_8u_C1 d_trg_C1(h_src_C1);

    Rectangle trg_rect(0, 0, (int)d_trg_C1.width(), (int)d_trg_C1.height(),
            (int)d_trg_C1.pitch());

    dim3 grid( (trg_rect.numElements() + X_STRIDE - 1) / X_STRIDE,
            trg_rect.height);
    dim3 tpb(X_STRIDE, 1);

    init_trg_kernel<<<grid, tpb>>>(d_src_C1.data(), d_trg_C1.data(),
            src_rect, trg_rect);
    
    CUDA_CALL(hipGetLastError());
    CUDA_CALL(hipDeviceSynchronize());

    d_trg_C1.copyTo(h_trg_C1.data(), h_trg_C1.pitch());

    npp::saveImage(args.output_name, h_trg_C1);

    nppiFree(h_src_C1.data());
    nppiFree(d_src_C1.data());
    nppiFree(h_trg_C1.data());
    nppiFree(d_trg_C1.data());

    exit (0);
}

